#include "hip/hip_runtime.h"
/*  Vector addition on the GPU: C = A + B  */
#include <stdio.h>
#include <stdlib.h>

// Max number of threads per block
#define SIZE 1024

// Device function (i.e. kernel)
__global__ void VecAdd(float * A, float * B, float * C, int N)
{

   int i = threadIdx.x;
   if ( i < N ) {
      C[i] = A[i] + B[i];
   }

}

// CPU version of the vector addition function
void vecAddCPU(float * A, float * B, float * C, int N)
{

   int i;
   for (i=0; i<N; i++)
   {
      C[i] = A[i] + B[i];
   }

}

// Function compares two 1d arrays
void compareVecs( float * vec1, float * vec2, int N )
{

   int i;
   int vecsEqual = 1;
   for (i=0; i<N; i++)
   {
      if ( abs (vec1[i] - vec2[i]) > 0.00001 )
      {
         printf("vectors not equal! i: %d  vec1[i]: %f  vec2[i]: %f\n",i,vec1[i],vec2[i]);
         vecsEqual = 0;
      }
   }
   if ( vecsEqual ) printf("GPU vector addition agrees with CPU version!\n");

}

/* Host function for filling vector (1d array) with 
   random numbers between -20.0 and 20.0 */
void fillOutVector( float * vec, int vec_length )
{

   time_t t;
   srand((unsigned) time(&t)); // initialize random number generator
   int i;
   for (i=0; i<vec_length; i++)
   {
      vec[i] = ( (float)rand() / (float)(RAND_MAX) ) * 40.0;
      vec[i] -= 20.0;
   }

}

// Host function for printing a vector (1d array)
void printVector( float * vec, int vec_length )
{
   int i;
   for (i=0; i<vec_length; i++) {
      printf("i: %d vec[i]: %f\n",i,vec[i]);
   }

}

// program execution begins here
int main( int argc, char ** argv )
{

   // CUDA events allow us to measure execution time of our kernel
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   // size_t is the appropriate type for bytes
   size_t vec_bytes = SIZE * sizeof(float);

   // host arrays
   float * h_A = (float *)malloc( vec_bytes );
   float * h_B = (float *)malloc( vec_bytes );
   float * h_C = (float *)malloc( vec_bytes );

   // fill array with random floats
   fillOutVector( h_A, SIZE );
   fillOutVector( h_B, SIZE );

   // device arrays
   float * d_A, * d_B, * d_C;
   hipError_t rc; // return code from cuda functions
   rc = hipMalloc(&d_A, vec_bytes);
   if ( rc ) printf("%s\n",hipGetErrorString(rc));
   hipMalloc(&d_B, vec_bytes);
   hipMalloc(&d_C, vec_bytes);

   // copy A and B to the device
   hipMemcpy(d_A, h_A, vec_bytes, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, vec_bytes, hipMemcpyHostToDevice);

   // dim3 is a 3-element struct with elements x, y, z (all ints)
   dim3 threadsPerBlock(SIZE); // SIZE x 1 x 1
   dim3 blocksPerGrid(1); // 1 x 1 x 1
   // launch vector addition kernel!
   hipEventRecord(start);
   VecAdd<<< blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, SIZE);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   printf("kernel time (ms) : %7.5f\n",milliseconds);

   // copy results to host
   hipMemcpy(h_C, d_C, vec_bytes, hipMemcpyDeviceToHost);
   //printVector( h_C, SIZE );

   // verify that we got correct results
   float * gold_C = (float *)malloc( vec_bytes );
   hipEventRecord(start);
   vecAddCPU( h_A, h_B, gold_C, SIZE );
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   printf("cpu function time (ms) : %7.5f\n",milliseconds);
   compareVecs( gold_C, h_C, SIZE );

   // free memory on device
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   // clean up timer variables
   hipEventDestroy(start);
   hipEventDestroy(stop);

   // free memory on host
   free(h_A);
   free(h_B);
   free(h_C);
   free(gold_C);

   return 0;
}