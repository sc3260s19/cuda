#include "hip/hip_runtime.h"
/*  Vector addition on the GPU: C = A + B  */
#include <stdio.h>
#include <stdlib.h>

// For consistency with GPU implementation
#define SIZE 1024

// CPU version of the vector addition function
void vecAddCPU(float * A, float * B, float * C, int N)
{

   int i;
   for (i=0; i<N; i++)
   {
      C[i] = A[i] + B[i];
   }

}

// Function compares two 1d arrays
void compareVecs( float * vec1, float * vec2, int N )
{

   int i;
   int vecsEqual = 1;
   for (i=0; i<N; i++)
   {
      if ( abs (vec1[i] - vec2[i]) > 0.00001 )
      {
         printf("vectors not equal! i: %d  vec1[i]: %f  vec2[i]: %f\n",i,vec1[i],vec2[i]);
         vecsEqual = 0;
      }
   }
   if ( vecsEqual ) printf("GPU vector addition agrees with CPU version!\n");

}

/* Host function for filling vector (1d array) with 
   random numbers between -20.0 and 20.0 */
void fillOutVector( float * vec, int vec_length )
{

   time_t t;
   srand((unsigned) time(&t)); // initialize random number generator
   int i;
   for (i=0; i<vec_length; i++)
   {
      vec[i] = ( (float)rand() / (float)(RAND_MAX) ) * 40.0;
      vec[i] -= 20.0;
   }

}

// Host function for printing a vector (1d array)
void printVector( float * vec, int vec_length )
{
   int i;
   for (i=0; i<vec_length; i++) {
      printf("i: %d vec[i]: %f\n",i,vec[i]);
   }

}

// program execution begins here
int main( int argc, char ** argv )
{

   size_t vec_bytes = SIZE * sizeof(float);

   // host arrays
   float * h_A = (float *)malloc( vec_bytes );
   float * h_B = (float *)malloc( vec_bytes );
   float * h_C = (float *)malloc( vec_bytes );

   // fill array with random floats
   fillOutVector( h_A, SIZE );
   fillOutVector( h_B, SIZE );

   // compute the sum of vector A and B on CPU
   float * gold_C = (float *)malloc( vec_bytes );
   vecAddCPU( h_A, h_B, gold_C, SIZE );

   // print result of vector addition
   printVector( gold_C, SIZE );

   /* compare two arrays; use this function to compare the array
      that was computed on the GPU to the one that was computed on the CPU */
   //compareVecs( gold_C, h_C, SIZE );

   // free memory on host
   free(h_A);
   free(h_B);
   free(h_C);
   free(gold_C);

   return 0;
}